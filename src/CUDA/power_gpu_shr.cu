#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "gputimer.h"

// Input Array Variables
float* h_MatA = NULL;
float* d_MatA = NULL;

// Output Array
float* h_VecV = NULL;
float* d_VecV = NULL;
float* h_VecW = NULL;
float* d_VecW = NULL;
float* h_NormW = NULL;
float* d_NormW = NULL;
float* h_lambda = NULL;
float* d_lambda = NULL;

// Variables to change
int GlobalSize = 5000;         // this is the dimension of the matrix, GlobalSize*GlobalSize
const int BLOCK_SIZE = 32;  // number of threads per block
const float EPS = 0.000005;    // tolerence of the error
int max_iteration = 100;       // the maximum iteration steps


// Functions
void Cleanup(void);
void InitOne(float*, int);
void UploadArray(float*, int);
void  Arguments(int, char**);
void checkCardVersion(void);

// Kernels

// The shared memory is limited for a block, instead of reading an entire row of matrix A or vector V from global memory to shared memory, 
// a square submatrix of A is shared by a block, the size of square submatrix is BLOCK_SIZE*BLOCK_SIZE; Thus, a for-loop is used to
// handle a multiplication of each row of Matrix A and vector V step by step. In eacg step, two subvectors with size BLOCK_SIZE is multiplied.
//*****************************************************************************************************************************************************/


__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N)
{
  // Block index
  int bx = blockIdx.x;

  // Thread index
  int tx = threadIdx.x;

  int aBegin = N * BLOCK_SIZE * bx;

  int aEnd   = aBegin + N - 1;
  int step  = BLOCK_SIZE;

  int bBegin = 0;//BLOCK_SIZE * bx;
  int bIndex=0;
  int aIndex =0;
  float Csub = 0;

  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += step, b += step)
  {

    __shared__ float As[BLOCK_SIZE*BLOCK_SIZE];

    __shared__ float bs[BLOCK_SIZE];
        

    for (int aa = 0; aa < BLOCK_SIZE;aa+= 1)
    {
      aIndex = a+tx+aa*N;
      if( aIndex < N*N)
        As[tx+aa*BLOCK_SIZE] = g_MatA[aIndex];
      else
        As[tx+aa*BLOCK_SIZE] = 0;
    }

    bIndex = b+tx;
    if(bIndex<N)   
      bs[tx] = g_VecV[bIndex];
    else
      bs[tx] = 0;

    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; ++k)
    {
      Csub += As[k+tx*BLOCK_SIZE] * bs[k];
    }//}
    __syncthreads();
  }

  g_VecW[ BLOCK_SIZE * bx + tx] = Csub;
}

/****************************************************
Normalizes vector W : W/norm(W)
****************************************************/
__global__ void FindNormW(float* g_VecW, float * g_NormW, int N)
{
  // shared memory size declared at kernel launch
  extern __shared__ float sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
  
  // For thread ids greater than data space
  if (globalid < N) {
    sdata[tid] =  g_VecW[globalid];
  }
  else {
    sdata[tid] = 0;  // Case of extra threads above N
  }

  // each thread loads one element from global to shared mem
  __syncthreads();

  sdata[tid] = sdata[tid] * sdata[tid];
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x / 2; s > 0; s = s >> 1) {
    if (tid < s) {
      sdata[tid] = sdata[tid] + sdata[tid+ s];
    }
    __syncthreads();
  }
  // atomic operations:
  if (tid == 0) atomicAdd(g_NormW,sdata[0]);

}

__global__ void NormalizeW(float* g_VecW, float * g_NormW, float* g_VecV, int N)
{
  // shared memory size declared at kernel launch
  extern __shared__ float sNormData[];
  unsigned int tid = threadIdx.x;
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;

  if(tid==0) sNormData[0] =  g_NormW[0];
  __syncthreads();

  // For thread ids greater than data space
  if (globalid < N) {
    g_VecV[globalid] = g_VecW[globalid]/sNormData[0];
  }

}

__global__ void ComputeLamda( float* g_VecV, float* g_VecW, float * g_Lamda,int N)
{
  // shared memory size declared at kernel launch
  extern __shared__ float sdataVW[];
  unsigned int tid = threadIdx.x;
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;

  // For thread ids greater than data space
  if (globalid < N) {
    sdataVW[tid] =  g_VecV[globalid] * g_VecW[globalid];
  }
  else {
    sdataVW[tid] = 0;  // Case of extra threads above N
  }

  // each thread loads one element from global to shared mem
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x / 2; s > 0; s = s >> 1) {
    if (tid < s) {
      sdataVW[tid] = sdataVW[tid] + sdataVW[tid+ s];
    }
    __syncthreads();
  }
  // atomic operations:
  if (tid == 0) atomicAdd(g_Lamda,sdataVW[0]);
}


// Host code
int main(int argc, char** argv)
{
  GpuTimer mem_timer, kernel_timer, total;
  float mem_time =0., kernel_time = 0.;
  
  Arguments(argc, argv);
		
  int N = GlobalSize;
  printf("%d X %d for threads/Block %d \n", N, N, BLOCK_SIZE);
  size_t vec_size = N * sizeof(float);
  size_t mat_size = N * N * sizeof(float);
  size_t norm_size = sizeof(float);
  
  // Allocate normalized value in host memory
  h_NormW = (float*)malloc(norm_size);
  // Allocate input matrix in host memory
  h_MatA = (float*)malloc(mat_size);
  // Allocate initial vector V in host memory
  h_VecV = (float*)malloc(vec_size);
  // Allocate W vector for computations
  h_VecW = (float*)malloc(vec_size);
  //Allocate lambda
  h_lambda = (float*)malloc(norm_size);


  // Set the kernel arguments
  int threadsPerBlock = BLOCK_SIZE;   
  int sharedMemSize = threadsPerBlock * threadsPerBlock * sizeof(float); // in per block, the memory is shared
  int sharedMemSize2 = threadsPerBlock * sizeof(float); //for norm
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  // Allocate matrix and vectors in device memory
  hipMalloc((void**)&d_MatA, mat_size); 
  hipMalloc((void**)&d_VecV, vec_size); 
  hipMalloc((void**)&d_VecW, vec_size); // This vector is only used by the device
  hipMalloc((void**)&d_NormW, norm_size); 
  hipMalloc((void**)&d_lambda, norm_size);

  // Initialize input matrix
  UploadArray(h_MatA, N);
  InitOne(h_VecV,N);
    
  /////////////////////////////////////////////////
  // This is the starting points of GPU
  checkCardVersion();
  printf("*************************************\n");
  printf("Power Method (shared mem) on GPU starts\n");
  
  total.Start(); //total runtime
  
  //Copy from host memory to device memory
  mem_timer.Start();
  hipMemcpy(d_MatA, h_MatA, mat_size, hipMemcpyHostToDevice);
  hipMemcpy(d_VecV, h_VecV, vec_size, hipMemcpyHostToDevice);
  mem_timer.Stop();
  mem_time += (mem_timer.Elapsed()*1e-3);

  kernel_timer.Start();
  Av_Product<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_MatA, d_VecV,
                                                 d_VecW, N);
  hipDeviceSynchronize(); //Needed, kind of barrier to sychronize all threads
  kernel_timer.Stop();
  kernel_time += (kernel_timer.Elapsed()*1e-3);
  
  //Power method loops
  float OldLamda = 0;
  
  for(int i=0; i<10;i++)
  {
    h_NormW[0]= 0;

   h_NormW[0]= 0;

    //Norm
    mem_timer.Start();
    hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
    mem_timer.Stop();
    mem_time += (mem_timer.Elapsed()*1e-3);

    kernel_timer.Start();
    FindNormW<<<blocksPerGrid, threadsPerBlock, sharedMemSize2>>>(d_VecW,d_NormW, N);
    hipDeviceSynchronize();
    kernel_timer.Stop();
    kernel_time += (kernel_timer.Elapsed()*1e-3);

    //Transfer to host & back
    mem_timer.Start();
    hipMemcpy(h_NormW,d_NormW, norm_size, hipMemcpyDeviceToHost);
    h_NormW[0] = sqrt(h_NormW[0]);    
    hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
    mem_timer.Stop();
    mem_time += (mem_timer.Elapsed()*1e-3);

    //Normalize
    kernel_timer.Start();
    NormalizeW<<<blocksPerGrid, threadsPerBlock,sharedMemSize>>>(d_VecW, d_NormW ,
                                                   d_VecV, N);
    hipDeviceSynchronize();
    kernel_timer.Stop();
    kernel_time += (kernel_timer.Elapsed()*1e-3);
    
    
    //AvProduct
    kernel_timer.Start();
    Av_Product<<<blocksPerGrid, threadsPerBlock,sharedMemSize>>>(d_MatA, d_VecV, d_VecW, N);
    hipDeviceSynchronize();
    kernel_timer.Stop();
    kernel_time += (kernel_timer.Elapsed()*1e-3);
    
    //Compute Lambda
    h_lambda[0]=0.;
    mem_timer.Start();
    hipMemcpy(d_lambda, h_lambda, norm_size, hipMemcpyHostToDevice);
    mem_timer.Stop();
    mem_time += (mem_timer.Elapsed()*1e-3);

    kernel_timer.Start();
    ComputeLamda<<<blocksPerGrid, threadsPerBlock,sharedMemSize>>>(d_VecV, d_VecW,
                                                     d_lambda , N);
    hipDeviceSynchronize();
    kernel_timer.Stop();
    kernel_time += (kernel_timer.Elapsed()*1e-3);

    
    mem_timer.Start();
    hipMemcpy(h_lambda, d_lambda, norm_size, hipMemcpyDeviceToHost);
    mem_timer.Stop();
    mem_time += (mem_timer.Elapsed()*1e-3);
    
    printf("GPU lamda at %d: %f \n", i, h_lambda[0]);
    //If residual is lass than epsilon break
    if(abs(OldLamda - h_lambda[0]) < EPS)
      break;
    OldLamda = h_lambda[0]; 	     
    
  }	
  ////////////////////////////////////////////////////////////
   

  total.Stop();
  printf("Power Method on GPU ends\n");
  printf("*************************************\n\n");
  printf("memtime,kerneltime,total\n");
  printf("%f,%f,%f\n",mem_time,kernel_time,total.Elapsed()*1e-3);

  Cleanup();
}

void Cleanup(void)
{
  // Free device memory
  if (d_MatA)
    hipFree(d_MatA);
  if (d_VecV)
    hipFree(d_VecV);
  if (d_VecW)
    hipFree(d_VecW);
  if (d_NormW)
    hipFree(d_NormW);
  if (d_lambda)
    hipFree(d_lambda);
		
  // Free host memory
  if (h_MatA)
    free(h_MatA);
  if (h_VecV)
    free(h_VecV);
  if (h_VecW)
    free(h_VecW);
  if (h_NormW)
    free(h_NormW);
  if(h_lambda)
    free(h_lambda);
    
  exit(0);
}

// Allocates an array with zero value.
void InitOne(float* data, int n)
{
  for (int i = 0; i < n; i++)
    data[i] = 0;
  data[0]=1;
}

void UploadArray(float* data, int n)
{
  int total = n*n;
  int value=1;
  for (int i = 0; i < total; i++)
  {
    data[i] = (int) (rand() % (int)(101));//1;//value;
    value ++; if(value>n) value =1;
    // data[i] = 1;
  }
}

// Obtain program arguments
void Arguments(int argc, char** argv)
{
  for (int i = 0; i < argc; ++i) 
  {
    if (strcmp(argv[i], "--size") == 0 || strcmp(argv[i], "-size") == 0)
    {
      GlobalSize = atoi(argv[i+1]);
      i = i + 1;
    }
    if (strcmp(argv[i], "--max_iteration") == 0 || strcmp(argv[i], "-max_iteration") == 0)
    {
      max_iteration = atoi(argv[i+1]);
      i = i + 1;
    }
  }
}


void checkCardVersion()
{
  hipDeviceProp_t prop;
   
  hipGetDeviceProperties(&prop, 0);
   
  printf("This GPU has major architecture %d, minor %d \n",prop.major,prop.minor);
  if(prop.major < 2)
  {
    fprintf(stderr,"Need compute capability 2 or higher.\n");
    exit(1);
  }
}
