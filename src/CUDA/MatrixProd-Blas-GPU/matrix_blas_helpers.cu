/* Some common routines for allocating Blas matrices,
 * filling them with some data and printing them.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>


void checkCublasStatus(hipblasStatus_t status, const char *error)
{
   if (status != HIPBLAS_STATUS_SUCCESS)
   {
      fprintf (stderr, "Cuda CUBLAS : %s\n",error);
      exit(EXIT_FAILURE);
   }
}

void checkCudaError(const char *errormsg)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess)
   {
      fprintf (stderr, "%s\n",errormsg);
      fprintf (stderr, "Cuda: %s\n",hipGetErrorString(error));
      exit(EXIT_FAILURE);
   }
}

float *initHostMatrix(int n, int m)
{
   float *ptr = 0;
   
   hipHostMalloc(&ptr, n * m * sizeof(float), hipHostMallocDefault); // rows x columns
   
   if (ptr == NULL)
   {
      fprintf(stderr,"Malloc for matrix on host failed !\n");
      exit(1);
   }
   
   return ptr;
}

// Note that we actually fill the TRANSPOSED matrix here
// as BLAS is Fortran based !!! 
void fillMatrix(float *a, int n, int m, int offset)
{  long x,y;
   
   for(y=0; y<m; y++)   // mind the order of the loops : this is 20x faster ...
     for(x=0; x<n; x++)
       a[y*n + x] = (float) x+y + offset;
}

// Note that we actually show the TRANSPOSED matrix here
// as BLAS is Fortran based !!! 
void showMatrix(const char *name, float *a, int n, int m)
{ 
   long x,y;
   
# if (DEBUG > 0)
   for(y=0; y<m; y++)
# else
   y = m - 1;
# endif
   {
# if (DEBUG > 1)
      for(x=0; x<n; x++)
# else
      x = n - 1;
# endif
      {
         printf("%s[%02ld][%02ld]=%6.2f  ",name,x,y,a[y*n + x]);
      }
      printf("\n");
   }
}

float *initCublasMatrix(int n, int m)
{
   float *ptr = 0;
   
   hipMalloc(&ptr, n*m * sizeof(float));
   checkCudaError("Malloc for matrix on device failed !");
   
   return ptr;
}

void copytoCublasMatrix(float *d_a, float *h_a, int n, int m)
{
   hipblasStatus_t status;
   
   status = hipblasSetVector(n*m, sizeof(float), h_a, 1, d_a, 1);
   checkCublasStatus(status," Matrix copy to device failed !");
}

void copyfromCublasMatrix(float *h_a, float *d_a, int n, int m)
{
   hipblasStatus_t status;
   
   status = hipblasGetVector(n*m, sizeof(float), d_a, 1, h_a, 1);
   checkCublasStatus(status,"Matrix copy from device failed !");
}

void freeCudaHost(float *a)
{
   hipHostFree(a);
   checkCudaError("Memory free error on host !");
}

void freeCudaDevice(float *a)
{
   hipFree(a);
   checkCudaError("Memory free error on device !");
}
